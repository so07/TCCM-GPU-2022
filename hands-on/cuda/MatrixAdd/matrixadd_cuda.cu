#include "hip/hip_runtime.h"
#include <stdio.h>

void  initMatrix(float *A, int n, int m, float c) {
  int i,j;
  for (i=0; i<n; i++)
    for (j=0; j<m; j++)
      A[i*m+j] = c;
}

__global__ void gpuMatAdd(float *A, float *B, float *C, int N, int M) 
{
  //----- insert CUDA code -----
  // define index
  int i = 
  int j = 
  //----------------------------

  //----- insert CUDA code -----
  // check that the thread is not out of the vector boundary

  //----------------------------

  int index = j*M + i; 
  //----- insert CUDA code -----
  // write the operation for the sum of vectors 

  //----------------------------
}

int main(int argc, char *argv[]) {

   // size of matrix
  const int N = 1000, M = 1000;

  // allocate memory on host
  float * A = (float *) malloc(N*M * sizeof(float));
  float * B = (float *) malloc(N*M * sizeof(float));
  float * C = (float *) malloc(N*M * sizeof(float));

  initMatrix((float *) A, N, M, 1.0);
  initMatrix((float *) B, N, M, 2.0);
  initMatrix((float *) C, N, M, 0.0);


  //----- insert CUDA code -----
  // allocate memory on device
  float *A_dev, *B_dev, *C_dev;

  //----------------------------

  //----- insert CUDA code -----
  // copy data from host to device

  //----------------------------

  dim3 block(32,32);
  dim3 grid(M/block.x + 1, N/block.y + 1);

  //----- insert CUDA code -----
  // define the execution configuration

  //----------------------------

  //----- insert CUDA code -----
  // copy data from device to host

  //----------------------------

  printf("%f %f %f\n", C[0], C[1], C[3]);

  //----- insert CUDA code -----
  // free resources on device

  //----------------------------

  // free resources on host
  free(A);
  free(B);
  free(C);

  return 0;
}

