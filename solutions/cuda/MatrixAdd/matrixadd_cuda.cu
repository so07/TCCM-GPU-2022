
#include <hip/hip_runtime.h>
#include <stdio.h>

void  initMatrix(float *A, int n, int m, float c) {
  int i,j;
  for (i=0; i<n; i++)
    for (j=0; j<m; j++)
      A[i*m+j] = c;
}

__global__ void gpuMatAdd(float *A, float *B, float *C, int N, int M) 
{
  //----- insert CUDA code -----
  // define index
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  //----------------------------

  //----- insert CUDA code -----
  // check that the thread is not out of the vector boundary
  if (i >= M || j >= N) return;
  //----------------------------

  int index = j*M + i; 
  //----- insert CUDA code -----
  // write the operation for the sum of vectors 
  C[index] = A[index] + B[index];
  //----------------------------
}

int main(int argc, char *argv[]) {

   // size of matrix
  const int N = 1000, M = 1000;

  // allocate memory on host
  float * A = (float *) malloc(N*M * sizeof(float));
  float * B = (float *) malloc(N*M * sizeof(float));
  float * C = (float *) malloc(N*M * sizeof(float));

  initMatrix((float *) A, N, M, 1.0);
  initMatrix((float *) B, N, M, 2.0);
  initMatrix((float *) C, N, M, 0.0);


  //----- insert CUDA code -----
  // allocate memory on device
  float *A_dev, *B_dev, *C_dev;
  hipMalloc((void **) &A_dev, M*N*sizeof(float));
  hipMalloc((void **) &B_dev, M*N*sizeof(float));
  hipMalloc((void **) &C_dev, M*N*sizeof(float));
  //----------------------------

  //----- insert CUDA code -----
  // copy data from host to device
  hipMemcpy(A_dev, A, M*N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(B_dev, B, M*N*sizeof(float), hipMemcpyHostToDevice);
  //----------------------------

  dim3 block(32,32);
  dim3 grid(M/block.x + 1, N/block.y + 1);

  //----- insert CUDA code -----
  // define the execution configuration
  gpuMatAdd<<<grid, block>>>(A_dev, B_dev, C_dev, N, M);
  //----------------------------

  //----- insert CUDA code -----
  // copy data from device to host
  hipMemcpy(C, C_dev, M*N*sizeof(float), hipMemcpyDeviceToHost);
  //----------------------------

  printf("%f %f %f\n", C[0], C[1], C[3]);

  //----- insert CUDA code -----
  // free resources on device
  hipFree(A_dev);
  hipFree(B_dev);
  hipFree(C_dev);
  //----------------------------

  // free resources on host
  free(A);
  free(B);
  free(C);

  return 0;
}

