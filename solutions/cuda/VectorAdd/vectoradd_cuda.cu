
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void  initVector(double *u, int n, double c) {
  int i;
  for (i=0; i<n; i++)
      u[i] = c;
}

__global__ void gpuVectAdd(double *u, double *v, double *z, int N) 
{
  //----- insert CUDA code -----
  // define index
  //int i = 
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  //----------------------------

  //----- insert CUDA code -----
  // check that the thread is not out of the vector boundary
  if (i >= N ) return;
  //----------------------------

  int index = i; 
  //----- insert CUDA code -----
  // write the operation for the sum of vectors 
  z[index] = u[index] + v[index];
  //----------------------------
}


int main(int argc, char *argv[]) {

  // size of vectors
  const int N = 1000;

  // allocate memory on host
  double * u = (double *) malloc(N * sizeof(double));
  double * v = (double *) malloc(N * sizeof(double));
  double * z = (double *) malloc(N * sizeof(double));

  initVector((double *) u, N, 1.0);
  initVector((double *) v, N, 2.0);
  initVector((double *) z, N, 0.0);

  //----- insert CUDA code -----
  // allocate memory on device
  double *u_dev, *v_dev, *z_dev;
  hipMalloc((void **) &u_dev, N*sizeof(double));
  hipMalloc((void **) &v_dev, N*sizeof(double));
  hipMalloc((void **) &z_dev, N*sizeof(double));
  //----------------------------

  //----- insert CUDA code -----
  // copy data from host to device
  hipMemcpy(u_dev, u, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(v_dev, v, N*sizeof(double), hipMemcpyHostToDevice);
  //----------------------------

  dim3 block(32);
  dim3 grid((N-1)/block.x + 1);

  //----- insert CUDA code -----
  // define the execution configuration
  //gpuVectAdd<<<...,...>>>(.....);
  gpuVectAdd<<<grid, block>>>(u_dev, v_dev, z_dev, N);
  //----------------------------

  //----- insert CUDA code -----
  // copy data from device to host
  hipMemcpy(z, z_dev, N*sizeof(double), hipMemcpyDeviceToHost);
  //----------------------------

  printf("%f %f %f\n", z[0], z[1], z[1]);

  //----- insert CUDA code -----
  // free resources on device
  hipFree(u_dev);
  hipFree(v_dev);
  hipFree(z_dev);
  //----------------------------

  // free resources on host
  free(u);
  free(v);
  free(z);

  return 0;
}

